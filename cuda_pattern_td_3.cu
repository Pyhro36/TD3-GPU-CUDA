#include "hip/hip_runtime.h"
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "wb.h"
#define BLOCK_SIZE 512 //@@ You can change this


#define wbCheck(stmt) \
do { \
    hipError_t err = stmt; \
    if (err != hipSuccess) { \
        wbLog(ERROR, "Failed to run stmt ", #stmt); \
        wbLog(ERROR, "Got CUDA error ... ", hipGetErrorString(err)); \
        return -1; \
    } \
} while (0)

__global__ void total(float *input, float *output, int len) {

    //@@ Load a segment of the input vector into shared memory
    int stride;

    __shared__ float partialSum[2 * BLOCK_SIZE];

    int tx = threadIdx.x;
    int bDim = blockDim.x;
    int bx = blockIdx.x;
    int block = bx * bDim;
    int start = 2 * block;
    int ii = start + tx;

    if (ii < len) {
        partialSum[tx] = input[ii];

        if ((ii + bDim) < len) {
            partialSum[bDim + tx] = input[ii + bDim];
        }
    }

    //@@ Traverse the reduction tree
    for (stride = bDim; stride > 0; stride /= 2) {
        __syncthreads();

        if ((tx < stride) && ((ii + stride) < len)) {
            partialSum[tx] += partialSum[tx + stride];
        }
    }

    //@@ Write the computed sum of the block to the output vector at the
    //@@ correct index

    __syncthreads();

    if (tx == 0) {
        output[bx] = partialSum[0];
    }
}

int main(int argc, char **argv) {
    int ii;
    wbArg_t args;
    float *hostInput; // The input 1D list
    float *hostOutput; // The output list
    float *deviceInput;
    float *deviceOutput;
    int numInputElements; // number of elements in the input list
    int numOutputElements; // number of elements in the output list

    args = wbArg_read(argc, argv);
    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);
    numOutputElements = numInputElements / (BLOCK_SIZE << 1);
    
    if (numInputElements % (BLOCK_SIZE << 1)) {
        numOutputElements++;
    }
    
    hostOutput = (float *)malloc(numOutputElements * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");
    wbLog(TRACE, "The number of input elements in the input is ", numInputElements);
    wbLog(TRACE, "The number of output elements in the input is ", numOutputElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    wbCheck(hipMalloc((void **)&deviceInput, numInputElements * sizeof(float)));
    wbCheck(hipMalloc((void **)&deviceOutput, numOutputElements * sizeof(float)));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    wbCheck(hipMemcpy((void *)deviceInput, (void *)hostInput, numInputElements * sizeof(float),
            hipMemcpyHostToDevice));
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    // gridDim = numOutputElements
    // blockDim = BLOCK_SIZE

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    total<<<numOutputElements, BLOCK_SIZE>>>(deviceInput, deviceOutput, numInputElements);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    wbCheck(hipMemcpy((void *)hostOutput, (void *)deviceOutput, numOutputElements * sizeof(float),
                       hipMemcpyDeviceToHost));
    wbTime_stop(Copy, "Copying output memory to the CPU");

    /********************************************************************
    * Reduce output vector on the host
    * NOTE: One could also perform the reduction of the output vector
    * recursively and support any size input. For simplicity, we do not
    * require that for this lab.
    ********************************************************************/
    for (ii = 1; ii < numOutputElements; ++ii) {
        hostOutput[0] += hostOutput[ii];
    }

    printf("sum = %.6f\n", hostOutput[0]);

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    wbCheck(hipFree((void *)deviceInput));
    wbCheck(hipFree((void *)deviceOutput));
    wbTime_stop(GPU, "Freeing GPU Memory");

    free(hostInput);
    free(hostOutput);
    return 0;
}

